#include "hip/hip_runtime.h"
#include "hist-equ-gpu.h"
#include <iostream>

void HistTest(PGM_IMG img_in) {
	int* h_hist = (int*) malloc(256 * sizeof(int));
	HistogramGPU(h_hist, img_in.img, img_in.h * img_in.w, 256);
	for(int i = 0; i < 256; i++) {
		std::cout<<h_hist[i]<<std::endl;
	}
}

void HistogramGPU(int * hist_out, unsigned char* img_in, int img_size, int nbr_bin) {
	int threadsPerBlock = 256;
	int numSMs = 192;

	// Initialize the histogram
	int* d_hist;
	hipMalloc(&d_hist, nbr_bin * sizeof(int));
	MemsetGPU<<<numSMs * 32, threadsPerBlock>>>(d_hist, nbr_bin);

	//Click the counter
	unsigned char* imgData;
	size_t imgDataSize = img_size * sizeof(unsigned char);
	hipMalloc(&imgData, imgDataSize);
	hipMemcpy(imgData, img_in, imgDataSize, hipMemcpyHostToDevice);
	HistogramGpuAction<<<numSMs * 32, threadsPerBlock>>>(d_hist, imgData, img_size);
	int* h_hist = (int*) malloc(256 * sizeof(int));

	//Copy back the memory
	hipMemcpy(hist_out, d_hist, nbr_bin * sizeof(int), hipMemcpyDeviceToHost);
}

__global__ void MemsetGPU(int* histOut, int nbr_bin) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nbr_bin; i += blockDim.x * gridDim.x) {
		histOut[i] = 0;
	}
}

__global__ void HistogramGpuAction(int * histOut, unsigned char * imgIn, int imgSize) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < imgSize; i += blockDim.x * gridDim.x) {
		atomicAdd(&histOut[imgIn[i]], 1);
	}
}
