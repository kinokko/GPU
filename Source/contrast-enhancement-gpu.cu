#include "hip/hip_runtime.h"
#include "hist-equ-gpu.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"


//YUV Part
PPM_IMG ContrastEnhancementGYUV(PPM_IMG img_in) {
	int img_size = img_in.w * img_in.h;
	int img_data_size = img_size * sizeof(unsigned char);

	//copy img data to device memory
	PPM_IMG d_img_rgb;
	d_img_rgb.h = img_in.h;
	d_img_rgb.w = img_in.w;
	hipMalloc(&d_img_rgb.img_r, img_data_size);
	hipMalloc(&d_img_rgb.img_g, img_data_size);
	hipMalloc(&d_img_rgb.img_b, img_data_size);
	hipMemcpy(d_img_rgb.img_r, img_in.img_r, img_data_size, hipMemcpyHostToDevice);
	hipMemcpy(d_img_rgb.img_g, img_in.img_g, img_data_size, hipMemcpyHostToDevice);
	hipMemcpy(d_img_rgb.img_b, img_in.img_b, img_data_size, hipMemcpyHostToDevice);

	// init a device memory for yuv img
	YUV_IMG d_img_yuv;
	d_img_yuv.h = img_in.h;
	d_img_yuv.w = img_in.w;
	hipMalloc(&d_img_yuv.img_y, img_data_size);
	hipMalloc(&d_img_yuv.img_u, img_data_size);
	hipMalloc(&d_img_yuv.img_v, img_data_size);

	
	RGB2YUV_G<<<BLOCKPERGRID, THREADSPERBLOCK>>>(d_img_yuv, d_img_rgb, img_size);


	//hist
	int nbr_bin = 256;
	int* d_hist;
	hipMalloc(&d_hist, sizeof(int)*nbr_bin);
	HistogramGPU(d_hist, d_img_yuv.img_y, img_size, nbr_bin);

	//hist_equ
	int* d_lut;
	hipMalloc(&d_lut, sizeof(int)*nbr_bin);
	int* d_min;
	hipMalloc(&d_min, sizeof(int));
	int* d_d;
	hipMalloc(&d_d, sizeof(int));
	ConstructLUTGPU(d_lut, d_hist, d_min, d_d, nbr_bin, img_size);

	unsigned char* proceed_img;
	hipMalloc(&proceed_img, img_data_size);
	HistogramEqualizationGPUAction<<<BLOCKPERGRID, THREADSPERBLOCK>>>(proceed_img, d_lut, d_img_yuv.img_y, img_size);
	hipFree(d_img_yuv.img_y);
	d_img_yuv.img_y = proceed_img;

	YUV2RGB_G<<<BLOCKPERGRID, THREADSPERBLOCK>>>(d_img_rgb, d_img_yuv, img_size);

	PPM_IMG h_img_rgb;
	h_img_rgb.h = img_in.h;
	h_img_rgb.w = img_in.w;
	h_img_rgb.img_r = (unsigned char*)malloc(img_data_size);
	h_img_rgb.img_g = (unsigned char*)malloc(img_data_size);
	h_img_rgb.img_b = (unsigned char*)malloc(img_data_size);
	hipMemcpy(h_img_rgb.img_r, d_img_rgb.img_r, img_data_size, hipMemcpyDeviceToHost);
	hipMemcpy(h_img_rgb.img_g, d_img_rgb.img_g, img_data_size, hipMemcpyDeviceToHost);
	hipMemcpy(h_img_rgb.img_b, d_img_rgb.img_b, img_data_size, hipMemcpyDeviceToHost);
	return h_img_rgb;
}

__global__ void RGB2YUV_G(YUV_IMG d_img_out, PPM_IMG d_img_in, int img_size) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < img_size; i += blockDim.x * gridDim.x) {
		unsigned char r = d_img_in.img_r[i];
		unsigned char g = d_img_in.img_g[i];
		unsigned char b = d_img_in.img_b[i];

		unsigned char y = (unsigned char)(0.299*r + 0.587*g + 0.114*b);
		unsigned char cb = (unsigned char)(-0.169*r - 0.331*g + 0.499*b + 128);
		unsigned char cr = (unsigned char)(0.499*r - 0.418*g - 0.0813*b + 128);

		d_img_out.img_y[i] = y;
		d_img_out.img_u[i] = cb;
		d_img_out.img_v[i] = cr;
	}
}

__global__ void YUV2RGB_G(PPM_IMG d_img_out, YUV_IMG d_img_in, int img_size) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < img_size; i += blockDim.x * gridDim.x) {
		int y = (int)d_img_in.img_y[i];
		int cb = (int)d_img_in.img_u[i] - 128;
		int cr = (int)d_img_in.img_v[i] - 128;

		int rt = (int)(y + 1.402*cr);
		int gt = (int)(y - 0.344*cb - 0.714*cr);
		int bt = (int)(y + 1.772*cb);

		d_img_out.img_r[i] = clip_rgb_gpu(rt);
		d_img_out.img_g[i] = clip_rgb_gpu(gt);
		d_img_out.img_b[i] = clip_rgb_gpu(bt);
	}
}

//End of YUV Part


//HSL Part
PPM_IMG ContrastEnhancementGHSL(PPM_IMG img_in){
	HSL_IMG hsl_med;
	PPM_IMG result;

	unsigned char * l_equ;
	int hist[256];

	hsl_med = rgb2hsl(img_in);
	l_equ = (unsigned char *)malloc(hsl_med.height*hsl_med.width*sizeof(unsigned char));

	histogram(hist, hsl_med.l, hsl_med.height * hsl_med.width, 256);
	histogram_equalization(l_equ, hsl_med.l, hist, hsl_med.width*hsl_med.height, 256);

	free(hsl_med.l);
	hsl_med.l = l_equ;

	result = hsl2rgb(hsl_med);
	free(hsl_med.h);
	free(hsl_med.s);
	free(hsl_med.l);
	return result;
}
//End of HSL Part

//Helper 
__device__ unsigned char clip_rgb_gpu(int x)
{
	if (x > 255)
		return 255;
	if (x < 0)
		return 0;

	return (unsigned char)x;
}